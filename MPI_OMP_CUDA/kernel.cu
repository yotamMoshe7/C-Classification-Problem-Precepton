#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "CudaMemPtr.h"

#include <stdio.h>

hipError_t trainWithCuda(CudaMemPtr* cudaMem, const double* weights,
	double w0, double* results);

void cudaAllocationAndCpy(CudaMemPtr* cudaMem, const double* coords, int numOfPoints, int dimensions);

void checkErrors(hipError_t cudaStatus, CudaMemPtr* cudaMem, const char* errorMessage);

void freeCudaMem(CudaMemPtr* cudaMem);


__global__ void trainKernel(double *coords, const double * weights, double w0,
	double *results, int dimensions, int numOfPoints, int pointsPerBlock);

__global__ void trainKernel(double *coords, const double * weights, double w0,
	double *results, int dimensions, int numOfPoints, int pointsPerBlock)
{
	int id = blockIdx.x * pointsPerBlock + threadIdx.x, i;

	if (blockIdx.x == gridDim.x - 1 && numOfPoints % blockDim.x <= threadIdx.x)
		return;

	results[id] = w0;

	for (i = 0; i < dimensions; i++)
	{
		results[id] += coords[id*dimensions + i] * weights[i];
	}
}

void cudaAllocationAndCpy(CudaMemPtr* cudaMem, const double* coords, int numOfPoints, int dimensions)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkErrors(cudaStatus, cudaMem,"hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	cudaStatus = hipMalloc((void**)&cudaMem->cudaCoords, numOfPoints * dimensions * sizeof(double));
	checkErrors(cudaStatus, cudaMem, "hipMalloc failed! cudaCoords");

	cudaStatus = hipMalloc((void**)&cudaMem->cudaWeights, dimensions * sizeof(double));
	checkErrors(cudaStatus, cudaMem,"hipMalloc failed! cudaWeights");

	cudaStatus = hipMalloc((void**)&cudaMem->cudaResults, numOfPoints * sizeof(double));
	checkErrors(cudaStatus, cudaMem, "hipMalloc failed! cudaWeights");

	cudaStatus = hipMemcpy(cudaMem->cudaCoords, coords, numOfPoints * dimensions * sizeof(double), hipMemcpyHostToDevice);
	checkErrors(cudaStatus, cudaMem, "hipMemcpy failed! cudaCoords");

	cudaMem->numOfPoints = numOfPoints;
	cudaMem->dimensions = dimensions;
}

hipError_t trainWithCuda(CudaMemPtr* cudaMem, const double* weights, 
	double w0, double* results)
{
	double* cudaCoords = cudaMem->cudaCoords;
	double* cudaWeights = cudaMem->cudaWeights;
	double* cudaResults = cudaMem->cudaResults;
	int numOfPoints = cudaMem->numOfPoints;
	int dimensions = cudaMem->dimensions;
	hipError_t cudaStatus;
	int numOfBlocks, remainBlock, pointsPerBlock;
	hipDeviceProp_t prop;
	cudaMem->w0 = w0;

	hipGetDeviceProperties(&prop, 0);

	cudaStatus = hipMemcpy(cudaMem->cudaWeights, weights, dimensions * sizeof(double), hipMemcpyHostToDevice);
	checkErrors(cudaStatus, cudaMem,
		"hipMemcpy failed! cudaWeights");

	pointsPerBlock = prop.maxThreadsPerBlock;
	numOfBlocks = numOfPoints / prop.maxThreadsPerBlock;
	remainBlock = numOfPoints % prop.maxThreadsPerBlock != 0;
	trainKernel<<<numOfBlocks + remainBlock, pointsPerBlock >>>(cudaMem->cudaCoords, cudaWeights, w0, cudaResults, dimensions, numOfPoints, pointsPerBlock);

	cudaStatus = hipGetLastError();
	checkErrors(cudaStatus, cudaMem,
		"classifyKernel failed!");

	cudaStatus = hipDeviceSynchronize();
	checkErrors(cudaStatus, cudaMem,
		"hipDeviceSynchronize failed after classifyKernel!");

	cudaStatus = hipMemcpy(results, cudaResults, numOfPoints * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors(cudaStatus, cudaMem,
		"hipMemcpy failed! results");

	return cudaStatus;
}

void freeCudaMem(CudaMemPtr* cudaMem)
{
	hipFree(cudaMem->cudaCoords);
	hipFree(cudaMem->cudaWeights);
	hipFree(cudaMem->cudaResults);
}

void checkErrors(hipError_t cudaStatus, 
	CudaMemPtr* cudaMem, const char* message)
{
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, message);
		hipFree(cudaMem->cudaCoords);
		hipFree(cudaMem->cudaWeights);
		hipFree(cudaMem->cudaResults);
	}
}
